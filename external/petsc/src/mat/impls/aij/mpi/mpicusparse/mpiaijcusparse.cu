#include "hip/hip_runtime.h"
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1

#include <petscconf.h>
#include <../src/mat/impls/aij/mpi/mpiaij.h> /*I "petscmat.h" I*/
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>
#include <../src/mat/impls/aij/mpi/mpicusparse/mpicusparsematimpl.h>
#include <thrust/advance.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <petscsf.h>

struct VecCUDAEquals {
  template <typename Tuple>
  __host__ __device__ void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<0>(t);
  }
};

static PetscErrorCode MatCOOStructDestroy_MPIAIJCUSPARSE(void **data)
{
  MatCOOStruct_MPIAIJ *coo = (MatCOOStruct_MPIAIJ *)*data;

  PetscFunctionBegin;
  PetscCall(PetscSFDestroy(&coo->sf));
  PetscCallCUDA(hipFree(coo->Ajmap1));
  PetscCallCUDA(hipFree(coo->Aperm1));
  PetscCallCUDA(hipFree(coo->Bjmap1));
  PetscCallCUDA(hipFree(coo->Bperm1));
  PetscCallCUDA(hipFree(coo->Aimap2));
  PetscCallCUDA(hipFree(coo->Ajmap2));
  PetscCallCUDA(hipFree(coo->Aperm2));
  PetscCallCUDA(hipFree(coo->Bimap2));
  PetscCallCUDA(hipFree(coo->Bjmap2));
  PetscCallCUDA(hipFree(coo->Bperm2));
  PetscCallCUDA(hipFree(coo->Cperm1));
  PetscCallCUDA(hipFree(coo->sendbuf));
  PetscCallCUDA(hipFree(coo->recvbuf));
  PetscCall(PetscFree(coo));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatSetPreallocationCOO_MPIAIJCUSPARSE(Mat mat, PetscCount coo_n, PetscInt coo_i[], PetscInt coo_j[])
{
  Mat_MPIAIJ          *mpiaij = (Mat_MPIAIJ *)mat->data;
  PetscBool            dev_ij = PETSC_FALSE;
  PetscMemType         mtype  = PETSC_MEMTYPE_HOST;
  PetscInt            *i, *j;
  PetscContainer       container_h;
  MatCOOStruct_MPIAIJ *coo_h, *coo_d;

  PetscFunctionBegin;
  PetscCall(PetscFree(mpiaij->garray));
  PetscCall(VecDestroy(&mpiaij->lvec));
#if defined(PETSC_USE_CTABLE)
  PetscCall(PetscHMapIDestroy(&mpiaij->colmap));
#else
  PetscCall(PetscFree(mpiaij->colmap));
#endif
  PetscCall(VecScatterDestroy(&mpiaij->Mvctx));
  mat->assembled     = PETSC_FALSE;
  mat->was_assembled = PETSC_FALSE;
  PetscCall(PetscGetMemType(coo_i, &mtype));
  if (PetscMemTypeDevice(mtype)) {
    dev_ij = PETSC_TRUE;
    PetscCall(PetscMalloc2(coo_n, &i, coo_n, &j));
    PetscCallCUDA(hipMemcpy(i, coo_i, coo_n * sizeof(PetscInt), hipMemcpyDeviceToHost));
    PetscCallCUDA(hipMemcpy(j, coo_j, coo_n * sizeof(PetscInt), hipMemcpyDeviceToHost));
  } else {
    i = coo_i;
    j = coo_j;
  }

  PetscCall(MatSetPreallocationCOO_MPIAIJ(mat, coo_n, i, j));
  if (dev_ij) PetscCall(PetscFree2(i, j));
  mat->offloadmask = PETSC_OFFLOAD_CPU;
  // Create the GPU memory
  PetscCall(MatSeqAIJCUSPARSECopyToGPU(mpiaij->A));
  PetscCall(MatSeqAIJCUSPARSECopyToGPU(mpiaij->B));

  // Copy the COO struct to device
  PetscCall(PetscObjectQuery((PetscObject)mat, "__PETSc_MatCOOStruct_Host", (PetscObject *)&container_h));
  PetscCall(PetscContainerGetPointer(container_h, (void **)&coo_h));
  PetscCall(PetscMalloc1(1, &coo_d));
  *coo_d = *coo_h; // do a shallow copy and then amend fields in coo_d

  PetscCall(PetscObjectReference((PetscObject)coo_d->sf)); // Since we destroy the sf in both coo_h and coo_d
  PetscCallCUDA(hipMalloc((void **)&coo_d->Ajmap1, (coo_h->Annz + 1) * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Aperm1, coo_h->Atot1 * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Bjmap1, (coo_h->Bnnz + 1) * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Bperm1, coo_h->Btot1 * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Aimap2, coo_h->Annz2 * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Ajmap2, (coo_h->Annz2 + 1) * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Aperm2, coo_h->Atot2 * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Bimap2, coo_h->Bnnz2 * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Bjmap2, (coo_h->Bnnz2 + 1) * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Bperm2, coo_h->Btot2 * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->Cperm1, coo_h->sendlen * sizeof(PetscCount)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->sendbuf, coo_h->sendlen * sizeof(PetscScalar)));
  PetscCallCUDA(hipMalloc((void **)&coo_d->recvbuf, coo_h->recvlen * sizeof(PetscScalar)));

  PetscCallCUDA(hipMemcpy(coo_d->Ajmap1, coo_h->Ajmap1, (coo_h->Annz + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Aperm1, coo_h->Aperm1, coo_h->Atot1 * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Bjmap1, coo_h->Bjmap1, (coo_h->Bnnz + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Bperm1, coo_h->Bperm1, coo_h->Btot1 * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Aimap2, coo_h->Aimap2, coo_h->Annz2 * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Ajmap2, coo_h->Ajmap2, (coo_h->Annz2 + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Aperm2, coo_h->Aperm2, coo_h->Atot2 * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Bimap2, coo_h->Bimap2, coo_h->Bnnz2 * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Bjmap2, coo_h->Bjmap2, (coo_h->Bnnz2 + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Bperm2, coo_h->Bperm2, coo_h->Btot2 * sizeof(PetscCount), hipMemcpyHostToDevice));
  PetscCallCUDA(hipMemcpy(coo_d->Cperm1, coo_h->Cperm1, coo_h->sendlen * sizeof(PetscCount), hipMemcpyHostToDevice));

  // Put the COO struct in a container and then attach that to the matrix
  PetscCall(PetscObjectContainerCompose((PetscObject)mat, "__PETSc_MatCOOStruct_Device", coo_d, MatCOOStructDestroy_MPIAIJCUSPARSE));
  PetscFunctionReturn(PETSC_SUCCESS);
}

__global__ static void MatPackCOOValues(const PetscScalar kv[], PetscCount nnz, const PetscCount perm[], PetscScalar buf[])
{
  PetscCount       i         = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscCount grid_size = gridDim.x * blockDim.x;
  for (; i < nnz; i += grid_size) buf[i] = kv[perm[i]];
}

__global__ static void MatAddLocalCOOValues(const PetscScalar kv[], InsertMode imode, PetscCount Annz, const PetscCount Ajmap1[], const PetscCount Aperm1[], PetscScalar Aa[], PetscCount Bnnz, const PetscCount Bjmap1[], const PetscCount Bperm1[], PetscScalar Ba[])
{
  PetscCount       i         = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscCount grid_size = gridDim.x * blockDim.x;
  for (; i < Annz + Bnnz; i += grid_size) {
    PetscScalar sum = 0.0;
    if (i < Annz) {
      for (PetscCount k = Ajmap1[i]; k < Ajmap1[i + 1]; k++) sum += kv[Aperm1[k]];
      Aa[i] = (imode == INSERT_VALUES ? 0.0 : Aa[i]) + sum;
    } else {
      i -= Annz;
      for (PetscCount k = Bjmap1[i]; k < Bjmap1[i + 1]; k++) sum += kv[Bperm1[k]];
      Ba[i] = (imode == INSERT_VALUES ? 0.0 : Ba[i]) + sum;
    }
  }
}

__global__ static void MatAddRemoteCOOValues(const PetscScalar kv[], PetscCount Annz2, const PetscCount Aimap2[], const PetscCount Ajmap2[], const PetscCount Aperm2[], PetscScalar Aa[], PetscCount Bnnz2, const PetscCount Bimap2[], const PetscCount Bjmap2[], const PetscCount Bperm2[], PetscScalar Ba[])
{
  PetscCount       i         = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscCount grid_size = gridDim.x * blockDim.x;
  for (; i < Annz2 + Bnnz2; i += grid_size) {
    if (i < Annz2) {
      for (PetscCount k = Ajmap2[i]; k < Ajmap2[i + 1]; k++) Aa[Aimap2[i]] += kv[Aperm2[k]];
    } else {
      i -= Annz2;
      for (PetscCount k = Bjmap2[i]; k < Bjmap2[i + 1]; k++) Ba[Bimap2[i]] += kv[Bperm2[k]];
    }
  }
}

static PetscErrorCode MatSetValuesCOO_MPIAIJCUSPARSE(Mat mat, const PetscScalar v[], InsertMode imode)
{
  Mat_MPIAIJ          *mpiaij = static_cast<Mat_MPIAIJ *>(mat->data);
  Mat                  A = mpiaij->A, B = mpiaij->B;
  PetscScalar         *Aa, *Ba;
  const PetscScalar   *v1 = v;
  PetscMemType         memtype;
  PetscContainer       container;
  MatCOOStruct_MPIAIJ *coo;

  PetscFunctionBegin;
  PetscCall(PetscObjectQuery((PetscObject)mat, "__PETSc_MatCOOStruct_Device", (PetscObject *)&container));
  PetscCheck(container, PetscObjectComm((PetscObject)mat), PETSC_ERR_PLIB, "Not found MatCOOStruct on this matrix");
  PetscCall(PetscContainerGetPointer(container, (void **)&coo));

  const auto &Annz   = coo->Annz;
  const auto &Annz2  = coo->Annz2;
  const auto &Bnnz   = coo->Bnnz;
  const auto &Bnnz2  = coo->Bnnz2;
  const auto &vsend  = coo->sendbuf;
  const auto &v2     = coo->recvbuf;
  const auto &Ajmap1 = coo->Ajmap1;
  const auto &Ajmap2 = coo->Ajmap2;
  const auto &Aimap2 = coo->Aimap2;
  const auto &Bjmap1 = coo->Bjmap1;
  const auto &Bjmap2 = coo->Bjmap2;
  const auto &Bimap2 = coo->Bimap2;
  const auto &Aperm1 = coo->Aperm1;
  const auto &Aperm2 = coo->Aperm2;
  const auto &Bperm1 = coo->Bperm1;
  const auto &Bperm2 = coo->Bperm2;
  const auto &Cperm1 = coo->Cperm1;

  PetscCall(PetscGetMemType(v, &memtype));
  if (PetscMemTypeHost(memtype)) { /* If user gave v[] in host, we need to copy it to device */
    PetscCallCUDA(hipMalloc((void **)&v1, coo->n * sizeof(PetscScalar)));
    PetscCallCUDA(hipMemcpy((void *)v1, v, coo->n * sizeof(PetscScalar), hipMemcpyHostToDevice));
  }

  if (imode == INSERT_VALUES) {
    PetscCall(MatSeqAIJCUSPARSEGetArrayWrite(A, &Aa)); /* write matrix values */
    PetscCall(MatSeqAIJCUSPARSEGetArrayWrite(B, &Ba));
  } else {
    PetscCall(MatSeqAIJCUSPARSEGetArray(A, &Aa)); /* read & write matrix values */
    PetscCall(MatSeqAIJCUSPARSEGetArray(B, &Ba));
  }

  PetscCall(PetscLogGpuTimeBegin());
  /* Pack entries to be sent to remote */
  if (coo->sendlen) {
    MatPackCOOValues<<<(coo->sendlen + 255) / 256, 256>>>(v1, coo->sendlen, Cperm1, vsend);
    PetscCallCUDA(hipPeekAtLastError());
  }

  /* Send remote entries to their owner and overlap the communication with local computation */
  PetscCall(PetscSFReduceWithMemTypeBegin(coo->sf, MPIU_SCALAR, PETSC_MEMTYPE_CUDA, vsend, PETSC_MEMTYPE_CUDA, v2, MPI_REPLACE));
  /* Add local entries to A and B */
  if (Annz + Bnnz > 0) {
    MatAddLocalCOOValues<<<(int)((Annz + Bnnz + 255) / 256), 256>>>(v1, imode, Annz, Ajmap1, Aperm1, Aa, Bnnz, Bjmap1, Bperm1, Ba);
    PetscCallCUDA(hipPeekAtLastError());
  }
  PetscCall(PetscSFReduceEnd(coo->sf, MPIU_SCALAR, vsend, v2, MPI_REPLACE));

  /* Add received remote entries to A and B */
  if (Annz2 + Bnnz2 > 0) {
    MatAddRemoteCOOValues<<<(int)((Annz2 + Bnnz2 + 255) / 256), 256>>>(v2, Annz2, Aimap2, Ajmap2, Aperm2, Aa, Bnnz2, Bimap2, Bjmap2, Bperm2, Ba);
    PetscCallCUDA(hipPeekAtLastError());
  }
  PetscCall(PetscLogGpuTimeEnd());

  if (imode == INSERT_VALUES) {
    PetscCall(MatSeqAIJCUSPARSERestoreArrayWrite(A, &Aa));
    PetscCall(MatSeqAIJCUSPARSERestoreArrayWrite(B, &Ba));
  } else {
    PetscCall(MatSeqAIJCUSPARSERestoreArray(A, &Aa));
    PetscCall(MatSeqAIJCUSPARSERestoreArray(B, &Ba));
  }
  if (PetscMemTypeHost(memtype)) PetscCallCUDA(hipFree((void *)v1));
  mat->offloadmask = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatMPIAIJGetLocalMatMerge_MPIAIJCUSPARSE(Mat A, MatReuse scall, IS *glob, Mat *A_loc)
{
  Mat             Ad, Ao;
  const PetscInt *cmap;

  PetscFunctionBegin;
  PetscCall(MatMPIAIJGetSeqAIJ(A, &Ad, &Ao, &cmap));
  PetscCall(MatSeqAIJCUSPARSEMergeMats(Ad, Ao, scall, A_loc));
  if (glob) {
    PetscInt cst, i, dn, on, *gidx;

    PetscCall(MatGetLocalSize(Ad, NULL, &dn));
    PetscCall(MatGetLocalSize(Ao, NULL, &on));
    PetscCall(MatGetOwnershipRangeColumn(A, &cst, NULL));
    PetscCall(PetscMalloc1(dn + on, &gidx));
    for (i = 0; i < dn; i++) gidx[i] = cst + i;
    for (i = 0; i < on; i++) gidx[i + dn] = cmap[i];
    PetscCall(ISCreateGeneral(PetscObjectComm((PetscObject)Ad), dn + on, gidx, PETSC_OWN_POINTER, glob));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatMPIAIJSetPreallocation_MPIAIJCUSPARSE(Mat B, PetscInt d_nz, const PetscInt d_nnz[], PetscInt o_nz, const PetscInt o_nnz[])
{
  Mat_MPIAIJ         *b              = (Mat_MPIAIJ *)B->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)b->spptr;
  PetscInt            i;

  PetscFunctionBegin;
  if (B->hash_active) {
    B->ops[0]      = b->cops;
    B->hash_active = PETSC_FALSE;
  }
  PetscCall(PetscLayoutSetUp(B->rmap));
  PetscCall(PetscLayoutSetUp(B->cmap));
  if (PetscDefined(USE_DEBUG) && d_nnz) {
    for (i = 0; i < B->rmap->n; i++) PetscCheck(d_nnz[i] >= 0, PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "d_nnz cannot be less than 0: local row %" PetscInt_FMT " value %" PetscInt_FMT, i, d_nnz[i]);
  }
  if (PetscDefined(USE_DEBUG) && o_nnz) {
    for (i = 0; i < B->rmap->n; i++) PetscCheck(o_nnz[i] >= 0, PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "o_nnz cannot be less than 0: local row %" PetscInt_FMT " value %" PetscInt_FMT, i, o_nnz[i]);
  }
#if defined(PETSC_USE_CTABLE)
  PetscCall(PetscHMapIDestroy(&b->colmap));
#else
  PetscCall(PetscFree(b->colmap));
#endif
  PetscCall(PetscFree(b->garray));
  PetscCall(VecDestroy(&b->lvec));
  PetscCall(VecScatterDestroy(&b->Mvctx));
  /* Because the B will have been resized we simply destroy it and create a new one each time */
  PetscCall(MatDestroy(&b->B));
  if (!b->A) {
    PetscCall(MatCreate(PETSC_COMM_SELF, &b->A));
    PetscCall(MatSetSizes(b->A, B->rmap->n, B->cmap->n, B->rmap->n, B->cmap->n));
  }
  if (!b->B) {
    PetscMPIInt size;
    PetscCallMPI(MPI_Comm_size(PetscObjectComm((PetscObject)B), &size));
    PetscCall(MatCreate(PETSC_COMM_SELF, &b->B));
    PetscCall(MatSetSizes(b->B, B->rmap->n, size > 1 ? B->cmap->N : 0, B->rmap->n, size > 1 ? B->cmap->N : 0));
  }
  PetscCall(MatSetType(b->A, MATSEQAIJCUSPARSE));
  PetscCall(MatSetType(b->B, MATSEQAIJCUSPARSE));
  PetscCall(MatBindToCPU(b->A, B->boundtocpu));
  PetscCall(MatBindToCPU(b->B, B->boundtocpu));
  PetscCall(MatSeqAIJSetPreallocation(b->A, d_nz, d_nnz));
  PetscCall(MatSeqAIJSetPreallocation(b->B, o_nz, o_nnz));
  PetscCall(MatCUSPARSESetFormat(b->A, MAT_CUSPARSE_MULT, cusparseStruct->diagGPUMatFormat));
  PetscCall(MatCUSPARSESetFormat(b->B, MAT_CUSPARSE_MULT, cusparseStruct->offdiagGPUMatFormat));
  B->preallocated  = PETSC_TRUE;
  B->was_assembled = PETSC_FALSE;
  B->assembled     = PETSC_FALSE;
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatMult_MPIAIJCUSPARSE(Mat A, Vec xx, Vec yy)
{
  Mat_MPIAIJ *a = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(VecScatterBegin(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->A->ops->mult)(a->A, xx, yy));
  PetscCall(VecScatterEnd(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->B->ops->multadd)(a->B, a->lvec, yy, yy));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatZeroEntries_MPIAIJCUSPARSE(Mat A)
{
  Mat_MPIAIJ *l = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(MatZeroEntries(l->A));
  PetscCall(MatZeroEntries(l->B));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatMultAdd_MPIAIJCUSPARSE(Mat A, Vec xx, Vec yy, Vec zz)
{
  Mat_MPIAIJ *a = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(VecScatterBegin(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->A->ops->multadd)(a->A, xx, yy, zz));
  PetscCall(VecScatterEnd(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->B->ops->multadd)(a->B, a->lvec, zz, zz));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatMultTranspose_MPIAIJCUSPARSE(Mat A, Vec xx, Vec yy)
{
  Mat_MPIAIJ *a = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall((*a->B->ops->multtranspose)(a->B, xx, a->lvec));
  PetscCall((*a->A->ops->multtranspose)(a->A, xx, yy));
  PetscCall(VecScatterBegin(a->Mvctx, a->lvec, yy, ADD_VALUES, SCATTER_REVERSE));
  PetscCall(VecScatterEnd(a->Mvctx, a->lvec, yy, ADD_VALUES, SCATTER_REVERSE));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatCUSPARSESetFormat_MPIAIJCUSPARSE(Mat A, MatCUSPARSEFormatOperation op, MatCUSPARSEStorageFormat format)
{
  Mat_MPIAIJ         *a              = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)a->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT_DIAG:
    cusparseStruct->diagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_MULT_OFFDIAG:
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparseStruct->diagGPUMatFormat    = format;
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  default:
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_SUP, "unsupported operation %d for MatCUSPARSEFormatOperation. Only MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_DIAG, and MAT_CUSPARSE_MULT_ALL are currently supported.", op);
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatSetFromOptions_MPIAIJCUSPARSE(Mat A, PetscOptionItems PetscOptionsObject)
{
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;
  Mat_MPIAIJ              *a              = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE      *cusparseStruct = (Mat_MPIAIJCUSPARSE *)a->spptr;

  PetscFunctionBegin;
  PetscOptionsHeadBegin(PetscOptionsObject, "MPIAIJCUSPARSE options");
  if (A->factortype == MAT_FACTOR_NONE) {
    PetscCall(PetscOptionsEnum("-mat_cusparse_mult_diag_storage_format", "sets storage format of the diagonal blocks of (mpi)aijcusparse gpu matrices for SpMV", "MatCUSPARSESetFormat", MatCUSPARSEStorageFormats, (PetscEnum)cusparseStruct->diagGPUMatFormat, (PetscEnum *)&format, &flg));
    if (flg) PetscCall(MatCUSPARSESetFormat(A, MAT_CUSPARSE_MULT_DIAG, format));
    PetscCall(PetscOptionsEnum("-mat_cusparse_mult_offdiag_storage_format", "sets storage format of the off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV", "MatCUSPARSESetFormat", MatCUSPARSEStorageFormats, (PetscEnum)cusparseStruct->offdiagGPUMatFormat, (PetscEnum *)&format, &flg));
    if (flg) PetscCall(MatCUSPARSESetFormat(A, MAT_CUSPARSE_MULT_OFFDIAG, format));
    PetscCall(PetscOptionsEnum("-mat_cusparse_storage_format", "sets storage format of the diagonal and off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV", "MatCUSPARSESetFormat", MatCUSPARSEStorageFormats, (PetscEnum)cusparseStruct->diagGPUMatFormat, (PetscEnum *)&format, &flg));
    if (flg) PetscCall(MatCUSPARSESetFormat(A, MAT_CUSPARSE_ALL, format));
  }
  PetscOptionsHeadEnd();
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatAssemblyEnd_MPIAIJCUSPARSE(Mat A, MatAssemblyType mode)
{
  Mat_MPIAIJ *mpiaij = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(MatAssemblyEnd_MPIAIJ(A, mode));
  if (mpiaij->lvec) PetscCall(VecSetType(mpiaij->lvec, VECSEQCUDA));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatDestroy_MPIAIJCUSPARSE(Mat A)
{
  Mat_MPIAIJ         *aij            = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)aij->spptr;

  PetscFunctionBegin;
  PetscCheck(cusparseStruct, PETSC_COMM_SELF, PETSC_ERR_COR, "Missing spptr");
  PetscCallCXX(delete cusparseStruct);
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJSetPreallocation_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJGetLocalMatMerge_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetPreallocationCOO_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetValuesCOO_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatCUSPARSESetFormat_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatConvert_mpiaijcusparse_hypre_C", NULL));
  PetscCall(MatDestroy_MPIAIJ(A));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* defines MatSetValues_MPICUSPARSE_Hash() */
#define TYPE AIJ
#define TYPE_AIJ
#define SUB_TYPE_CUSPARSE
#include "../src/mat/impls/aij/mpi/mpihashmat.h"
#undef TYPE
#undef TYPE_AIJ
#undef SUB_TYPE_CUSPARSE

static PetscErrorCode MatSetUp_MPI_HASH_CUSPARSE(Mat A)
{
  Mat_MPIAIJ         *b              = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)b->spptr;

  PetscFunctionBegin;
  PetscCall(MatSetUp_MPI_Hash(A));
  PetscCall(MatCUSPARSESetFormat(b->A, MAT_CUSPARSE_MULT, cusparseStruct->diagGPUMatFormat));
  PetscCall(MatCUSPARSESetFormat(b->B, MAT_CUSPARSE_MULT, cusparseStruct->offdiagGPUMatFormat));
  A->preallocated = PETSC_TRUE;
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_INTERN PetscErrorCode MatConvert_MPIAIJ_MPIAIJCUSPARSE(Mat B, MatType, MatReuse reuse, Mat *newmat)
{
  Mat_MPIAIJ *a;
  Mat         A;

  PetscFunctionBegin;
  PetscCall(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
  if (reuse == MAT_INITIAL_MATRIX) PetscCall(MatDuplicate(B, MAT_COPY_VALUES, newmat));
  else if (reuse == MAT_REUSE_MATRIX) PetscCall(MatCopy(B, *newmat, SAME_NONZERO_PATTERN));
  A             = *newmat;
  A->boundtocpu = PETSC_FALSE;
  PetscCall(PetscFree(A->defaultvectype));
  PetscCall(PetscStrallocpy(VECCUDA, &A->defaultvectype));

  a = (Mat_MPIAIJ *)A->data;
  if (a->A) PetscCall(MatSetType(a->A, MATSEQAIJCUSPARSE));
  if (a->B) PetscCall(MatSetType(a->B, MATSEQAIJCUSPARSE));
  if (a->lvec) PetscCall(VecSetType(a->lvec, VECSEQCUDA));

  if (reuse != MAT_REUSE_MATRIX && !a->spptr) PetscCallCXX(a->spptr = new Mat_MPIAIJCUSPARSE);

  A->ops->assemblyend           = MatAssemblyEnd_MPIAIJCUSPARSE;
  A->ops->mult                  = MatMult_MPIAIJCUSPARSE;
  A->ops->multadd               = MatMultAdd_MPIAIJCUSPARSE;
  A->ops->multtranspose         = MatMultTranspose_MPIAIJCUSPARSE;
  A->ops->setfromoptions        = MatSetFromOptions_MPIAIJCUSPARSE;
  A->ops->destroy               = MatDestroy_MPIAIJCUSPARSE;
  A->ops->zeroentries           = MatZeroEntries_MPIAIJCUSPARSE;
  A->ops->productsetfromoptions = MatProductSetFromOptions_MPIAIJBACKEND;
  A->ops->setup                 = MatSetUp_MPI_HASH_CUSPARSE;

  PetscCall(PetscObjectChangeTypeName((PetscObject)A, MATMPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJGetLocalMatMerge_C", MatMPIAIJGetLocalMatMerge_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJSetPreallocation_C", MatMPIAIJSetPreallocation_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatCUSPARSESetFormat_C", MatCUSPARSESetFormat_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetPreallocationCOO_C", MatSetPreallocationCOO_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetValuesCOO_C", MatSetValuesCOO_MPIAIJCUSPARSE));
#if defined(PETSC_HAVE_HYPRE)
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatConvert_mpiaijcusparse_hypre_C", MatConvert_AIJ_HYPRE));
#endif
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_EXTERN PetscErrorCode MatCreate_MPIAIJCUSPARSE(Mat A)
{
  PetscFunctionBegin;
  PetscCall(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
  PetscCall(MatCreate_MPIAIJ(A));
  PetscCall(MatConvert_MPIAIJ_MPIAIJCUSPARSE(A, MATMPIAIJCUSPARSE, MAT_INPLACE_MATRIX, &A));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*@
  MatCreateAIJCUSPARSE - Creates a sparse matrix in `MATAIJCUSPARSE` (compressed row) format
  (the default parallel PETSc format).  This matrix will ultimately pushed down
  to NVIDIA GPUs and use the CuSPARSE library for calculations.

  Collective

  Input Parameters:
+ comm  - MPI communicator, set to `PETSC_COMM_SELF`
. m     - number of local rows (or `PETSC_DECIDE` to have calculated if `M` is given)
           This value should be the same as the local size used in creating the
           y vector for the matrix-vector product y = Ax.
. n     - This value should be the same as the local size used in creating the
       x vector for the matrix-vector product y = Ax. (or PETSC_DECIDE to have
       calculated if `N` is given) For square matrices `n` is almost always `m`.
. M     - number of global rows (or `PETSC_DETERMINE` to have calculated if `m` is given)
. N     - number of global columns (or `PETSC_DETERMINE` to have calculated if `n` is given)
. d_nz  - number of nonzeros per row in DIAGONAL portion of local submatrix
           (same value is used for all local rows)
. d_nnz - array containing the number of nonzeros in the various rows of the
           DIAGONAL portion of the local submatrix (possibly different for each row)
           or `NULL`, if `d_nz` is used to specify the nonzero structure.
           The size of this array is equal to the number of local rows, i.e `m`.
           For matrices you plan to factor you must leave room for the diagonal entry and
           put in the entry even if it is zero.
. o_nz  - number of nonzeros per row in the OFF-DIAGONAL portion of local
           submatrix (same value is used for all local rows).
- o_nnz - array containing the number of nonzeros in the various rows of the
           OFF-DIAGONAL portion of the local submatrix (possibly different for
           each row) or `NULL`, if `o_nz` is used to specify the nonzero
           structure. The size of this array is equal to the number
           of local rows, i.e `m`.

  Output Parameter:
. A - the matrix

  Level: intermediate

  Notes:
  It is recommended that one use the `MatCreate()`, `MatSetType()` and/or `MatSetFromOptions()`,
  MatXXXXSetPreallocation() paradigm instead of this routine directly.
  [MatXXXXSetPreallocation() is, for example, `MatSeqAIJSetPreallocation()`]

  The AIJ format, also called the
  compressed row storage), is fully compatible with standard Fortran
  storage.  That is, the stored row and column indices can begin at
  either one (as in Fortran) or zero.

.seealso: [](ch_matrices), `Mat`, `MATAIJCUSPARSE`, `MatCreate()`, `MatCreateAIJ()`, `MatSetValues()`, `MatSeqAIJSetColumnIndices()`, `MatCreateSeqAIJWithArrays()`, `MATMPIAIJCUSPARSE`
@*/
PetscErrorCode MatCreateAIJCUSPARSE(MPI_Comm comm, PetscInt m, PetscInt n, PetscInt M, PetscInt N, PetscInt d_nz, const PetscInt d_nnz[], PetscInt o_nz, const PetscInt o_nnz[], Mat *A)
{
  PetscMPIInt size;

  PetscFunctionBegin;
  PetscCall(MatCreate(comm, A));
  PetscCall(MatSetSizes(*A, m, n, M, N));
  PetscCallMPI(MPI_Comm_size(comm, &size));
  if (size > 1) {
    PetscCall(MatSetType(*A, MATMPIAIJCUSPARSE));
    PetscCall(MatMPIAIJSetPreallocation(*A, d_nz, d_nnz, o_nz, o_nnz));
  } else {
    PetscCall(MatSetType(*A, MATSEQAIJCUSPARSE));
    PetscCall(MatSeqAIJSetPreallocation(*A, d_nz, d_nnz));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*MC
   MATAIJCUSPARSE - A matrix type to be used for sparse matrices; it is as same as `MATMPIAIJCUSPARSE`.

   A matrix type whose data resides on NVIDIA GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on NVIDIA GPUs using the CuSPARSE library.

   This matrix type is identical to `MATSEQAIJCUSPARSE` when constructed with a single process communicator,
   and `MATMPIAIJCUSPARSE` otherwise.  As a result, for single process communicators,
   `MatSeqAIJSetPreallocation()` is supported, and similarly `MatMPIAIJSetPreallocation()` is supported
   for communicators controlling multiple processes.  It is recommended that you call both of
   the above preallocation routines for simplicity.

   Options Database Keys:
+  -mat_type mpiaijcusparse - sets the matrix type to `MATMPIAIJCUSPARSE`
.  -mat_cusparse_storage_format csr - sets the storage format of diagonal and off-diagonal matrices. Other options include ell (ellpack) or hyb (hybrid).
.  -mat_cusparse_mult_diag_storage_format csr - sets the storage format of diagonal matrix. Other options include ell (ellpack) or hyb (hybrid).
-  -mat_cusparse_mult_offdiag_storage_format csr - sets the storage format of off-diagonal matrix. Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

.seealso: [](ch_matrices), `Mat`, `MatCreateAIJCUSPARSE()`, `MATSEQAIJCUSPARSE`, `MATMPIAIJCUSPARSE`, `MatCreateSeqAIJCUSPARSE()`, `MatCUSPARSESetFormat()`, `MatCUSPARSEStorageFormat`, `MatCUSPARSEFormatOperation`
M*/

/*MC
   MATMPIAIJCUSPARSE - A matrix type to be used for sparse matrices; it is as same as `MATAIJCUSPARSE`.

  Level: beginner

.seealso: [](ch_matrices), `Mat`, `MATAIJCUSPARSE`, `MATSEQAIJCUSPARSE`
M*/
