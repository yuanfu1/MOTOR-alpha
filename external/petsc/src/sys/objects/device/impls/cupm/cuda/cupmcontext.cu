#include "../cupmcontext.hpp" /*I "petscdevice.h" I*/

using namespace Petsc::device::cupm;

PetscErrorCode PetscDeviceContextCreate_CUDA(PetscDeviceContext dctx)
{
  static constexpr auto cuda_context = CUPMContextCuda();

  PetscFunctionBegin;
  PetscCall(cuda_context.initialize(dctx->device));
  dctx->data = new PetscDeviceContext_(CUDA);
  *dctx->ops = cuda_context.ops;
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Management of CUBLAS and CUSOLVER handles */
PetscErrorCode PetscCUBLASGetHandle(hipblasHandle_t *handle)
{
  PetscDeviceContext dctx;

  PetscFunctionBegin;
  PetscAssertPointer(handle, 1);
  PetscCall(PetscDeviceContextGetCurrentContextAssertType_Internal(&dctx, PETSC_DEVICE_CUDA));
  PetscCall(PetscDeviceContextGetBLASHandle_Internal(dctx, handle));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscCUSOLVERDnGetHandle(hipsolverHandle_t *handle)
{
  PetscDeviceContext dctx;

  PetscFunctionBegin;
  PetscAssertPointer(handle, 1);
  PetscCall(PetscDeviceContextGetCurrentContextAssertType_Internal(&dctx, PETSC_DEVICE_CUDA));
  PetscCall(PetscDeviceContextGetSOLVERHandle_Internal(dctx, handle));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscGetCurrentCUDAStream(hipStream_t *stream)
{
  PetscDeviceContext dctx;
  void              *handle;

  PetscFunctionBegin;
  PetscAssertPointer(stream, 1);
  PetscCall(PetscDeviceContextGetCurrentContextAssertType_Internal(&dctx, PETSC_DEVICE_CUDA));
  PetscCall(PetscDeviceContextGetStreamHandle(dctx, &handle));
  *stream = *(hipStream_t *)handle;
  PetscFunctionReturn(PETSC_SUCCESS);
}
